#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "bit_attack.h"
}
__global__ void sign_attacker_kernel(float *x, float *grad, int n, float epsilon)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    if(index >= n) return;
    if(*grad > .000001) *x -= epsilon;
    else if(*grad < -0.000001) *x += epsilon;
    else *x = *x;
}

void sign_attacker_gpu(float *x_gpu, int *loc, int topk, float *grad_gpu, float epsilon)
{
    //printf("attack sign gpu!\n");
    int i, idx;
    for(i = 0; i < topk; i++){
        idx = loc[i];
        sign_attacker_kernel<<<1, 1>>>(&x_gpu[idx], &grad_gpu[idx], 1, epsilon);
    }
}
__global__ void sign_delete_kernel(float *x, float *grad, int n, float epsilon)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    if(index >= n) return;
    if(*grad > .000001) *x += epsilon;
    else if(*grad < -0.000001) *x -= epsilon;
    else *x = *x;
}

void sign_delete_gpu(float *x_gpu, int *loc, int topk, float *grad_gpu, float epsilon)
{
    //printf("delete sign gpu!\n");
    int i, idx;
    for(i = 0; i < topk; i++){
        idx = loc[i];
        sign_delete_kernel<<<1, 1>>>(&x_gpu[idx], &grad_gpu[idx], 1, epsilon);
    }
}

void bit_flip_attacker_gpu(attack_args a)
{
    int idx = a.mloss_loc[a.layer_idx][a.k_idx];
    float *x = a.x_gpu[a.layer_idx];
    int bit_idx = a.bit_idx;
    inject_noise_float_onebit_gpu(x, idx, bit_idx);
}


